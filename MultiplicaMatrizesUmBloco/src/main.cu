#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "lib.c"

__global__ void kernelMulti(int * a, int * b, int * c, int tamanho){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int value = 0;
	int indice = tx * blockDim.x + ty;

	if(indice <= tamanho){
		for(int i = 0; i< blockDim.x; i++){
			int x = a[tx * blockDim.x + i] ;
			int y = b[i * blockDim.x + ty] ;
			value += x * y;
		}
		c[indice] = value;
	}
}

int main(){
	dim3 a(2,3),b(3,4),c;
	c.x = a.x;
	c.y = b.y;

	int t_a = a.x * a.y;
	int t_b = b.x * b.y;
	int t_c = c.x * c.y;

	int tamA = t_a * sizeof(int);
	int tamB = t_b * sizeof(int);
	int tamC = t_c * sizeof(int);

	int * h_a = (int *) malloc(tamA);
	int * h_b = (int *) malloc(tamB);
	int * h_c = (int *) malloc(tamC);
	int * d_a;
	int * d_b;
	int * d_c;

	hipMalloc(&d_a, tamA);
	hipMalloc(&d_b, tamB);
	hipMalloc(&d_c, tamC);

	populaVetor(h_a, t_a);
	populaVetor(h_b, t_b);

	exibeVetor(h_a, t_a);
	printf("------------------\n");
	exibeVetor(h_b, t_b);
	printf("------------------\n");

	hipMemcpy(d_a,h_a,tamA,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,tamB,hipMemcpyHostToDevice);

	dim3 grid(1,1,1), block(c.x,c.y,1);
	kernelMulti<<<grid, block>>>(d_a, d_b, d_c, t_c);

	hipMemcpy(h_c,d_c, tamC, hipMemcpyDeviceToHost);

	printf("\n");
	exibeVetor(h_c, t_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	hipDeviceReset();

	return 0;
}
