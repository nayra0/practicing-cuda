#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "lib.c"

__global__ void kernelMulti(int * a, int * b, int * c, int tamanho, int dimA, int dimB, int dimX){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int value = 0;
	int indice = tx * blockDim.y + ty;

	if(indice <= tamanho){
		for(int i = 0; i< dimX; i++){
			int x = a[tx * dimA + i] ;
			int y = b[i * dimB + ty] ;
			value += x * y;
		}
		c[indice] = value;
	}
}

int main(){
	int dimComum = 3;
	dim3 a(2,dimComum),b(dimComum,4),c;
	c.x = a.x;
	c.y = b.y;

	int t_a = a.x * a.y;
	int t_b = b.x * b.y;
	int t_c = c.x * c.y;

	int tamA = t_a * sizeof(int);
	int tamB = t_b * sizeof(int);
	int tamC = t_c * sizeof(int);

	int * h_a = (int *) malloc(tamA);
	int * h_b = (int *) malloc(tamB);
	int * h_c = (int *) malloc(tamC);
	int * d_a;
	int * d_b;
	int * d_c;

	hipMalloc(&d_a, tamA);
	hipMalloc(&d_b, tamB);
	hipMalloc(&d_c, tamC);

	populaVetor(h_a, t_a);
	populaVetor(h_b, t_b);

	exibeVetor(h_a, t_a);
	printf("------------------\n");
	exibeVetor(h_b, t_b);
	printf("------------------\n");

	hipMemcpy(d_a,h_a,tamA,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,tamB,hipMemcpyHostToDevice);

	dim3 grid(1,1,1), block(c.x,c.y,1);
	kernelMulti<<<grid, block>>>(d_a, d_b, d_c, t_c, a.y, b.y, dimComum);

	hipMemcpy(h_c,d_c, tamC, hipMemcpyDeviceToHost);

	printf("\n");
	exibeVetor(h_c, t_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	hipDeviceReset();

	return 0;
}
