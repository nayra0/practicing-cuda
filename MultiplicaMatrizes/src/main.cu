#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "lib.c"

/**
 * Para essa implementação tem que ser verdade que grid.x > grid.y e block.y > block.x
 */
__global__ void kernelMulti(int * a, int * b, int * c, int tamanho, int dimA, int dimB, int dimX){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int row = bx * blockDim.x + tx;
	int col = by * blockDim.y + ty;

	int value = 0;
	int indice = gridDim.y * blockDim.y * row +col;

	if(indice <= tamanho){

		for(int i = 0; i< dimX; i++){
			int x = a[row * dimA + i] ;
			int y = b[i * dimB + col] ;
			value += x * y;
		}
		c[indice] = value;
	}
}

int main(){
	int dimComum = 3;
	dim3 a(4,dimComum),b(dimComum,3),c;
	c.x = a.x;
	c.y = b.y;

	int t_a = a.x * a.y;
	int t_b = b.x * b.y;
	int t_c = c.x * c.y;

	int tamA = t_a * sizeof(int);
	int tamB = t_b * sizeof(int);
	int tamC = t_c * sizeof(int);

	int * h_a = (int *) malloc(tamA);
	int * h_b = (int *) malloc(tamB);
	int * h_c = (int *) malloc(tamC);
	int * d_a;
	int * d_b;
	int * d_c;

	hipMalloc(&d_a, tamA);
	hipMalloc(&d_b, tamB);
	hipMalloc(&d_c, tamC);

	populaVetor(h_a, t_a);
	populaVetor(h_b, t_b);

	exibeMatriz(h_a, t_a, a.y);
	printf("------------------\n");
	exibeMatriz(h_b, t_b, b.y);
	printf("------------------\n");

	hipMemcpy(d_a,h_a,tamA,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,tamB,hipMemcpyHostToDevice);

	dim3 grid(2,1,1), block(2,3,1);
	kernelMulti<<<grid, block>>>(d_a, d_b, d_c, t_c, a.y, b.y, dimComum);

	hipMemcpy(h_c,d_c, tamC, hipMemcpyDeviceToHost);

	printf("\n");
	exibeMatriz(h_c, t_c, c.y);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	hipDeviceReset();

	return 0;
}
