#include<stdio.h>
#include<hip/hip_runtime.h>


#define MAX 10

void teste(){
	printf("hello world!");
}

void populaVetor(int * x, int tamanho){
	int i;
	for(i = 0; i < tamanho; i++){
		x[i] = rand() % tamanho;
	}
}

void exibeVetor(int * x, int tamanho){
	int i;
	for(i = 0; i < tamanho; i++){
		printf("%d : %d\n", i, x[i]);
	}
}


__global__ void kernelSoma(int * J, int * K, int * L, int tamanho){
	int indice = blockDim.x * blockIdx.x + threadIdx.x;
	if(indice < tamanho){
		L[indice] = J[indice] + K[indice];
	}
}

int main(){
//	teste();

	int tamanhoBytes = MAX * sizeof(int);

	int * h_J = (int * )malloc(tamanhoBytes);
	int * h_K = (int * )malloc(tamanhoBytes);
	int * h_L = (int * )malloc(tamanhoBytes);
	int * d_J;
	int * d_K;
	int * d_L;

	hipMalloc((void **) &d_J, tamanhoBytes);
	hipMalloc((void **) &d_K, tamanhoBytes);
	hipMalloc((void **) &d_L, tamanhoBytes);

	populaVetor(h_J, MAX);
	populaVetor(h_K, MAX);

	exibeVetor(h_J, MAX);
	exibeVetor(h_K, MAX);

	hipMemcpy(d_J, h_J, tamanhoBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_K, h_K, tamanhoBytes, hipMemcpyHostToDevice);

	dim3 grid, block;
	grid.x = 2;
	grid.y = 1;
	grid.z = 1;
	block.x = 5;
	block.y = 1;
	block.z = 1;

	kernelSoma<<<grid, block>>>(d_J, d_K, d_L, MAX);


	hipMemcpy(h_L, d_L, tamanhoBytes, hipMemcpyDeviceToHost);

	exibeVetor(h_L, MAX);

	hipFree(d_J);
	hipFree(d_K);
	hipFree(d_L);

	free(h_J);
	free(h_K);
	free(h_L);

	hipDeviceReset();
	return 0;
}



